#include <iostream>
#include "../../utils.cuh"
#include "naive.cuh"

static int M, N, K;

int main(int argc, char* argv[]) {
    M = std::stoi(argv[1]);
    N = std::stoi(argv[2]);
    K = std::stoi(argv[3]);

    hipStream_t stream;
    hipStreamCreate(&stream);
    
    float *d_A, *d_B, *d_C;
    auto A = new float[M * K];
    auto B = new float[K * N];
    fill_increment(A, M * K);
    fill_increment(B, K * N);
    
    auto C = new float[M * N];
    auto A_size = sizeof(float) * M * K;
    auto B_size = sizeof(float) * K * N;
    auto C_size = sizeof(float) * M * N;
    hipMallocAsync((void**)&d_A, A_size, stream);
    hipMallocAsync((void**)&d_B, B_size, stream);
    hipMallocAsync((void**)&d_C, C_size, stream);
    hipMemcpyAsync(d_A, A, A_size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_B, B, B_size, hipMemcpyHostToDevice, stream);
    
    dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32), 1);  // avoid grid & block z-axis for now
    dim3 blockDim(32, 32, 1);
    naiveKernel<<<gridDim, blockDim, 0, stream>>>(M, N, K, 1.0f, d_A, d_B, 0.0f, d_C);
    CUDA_CHECK(hipGetLastError());
    hipMemcpyAsync(C, d_C, C_size, hipMemcpyDeviceToHost, stream);
    // hipStreamSynchronize(stream);
    
    print(C, M, N);
    
    delete[] A;
    delete[] B;
    delete[] C;
    hipFreeAsync(d_A, stream);
    hipFreeAsync(d_B, stream);
    hipFreeAsync(d_C, stream);
    hipStreamDestroy(stream);
    return 0;
}